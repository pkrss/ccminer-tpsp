#include "hip/hip_runtime.h"

/*
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""



#include <stdint.h>
#include <memory.h>
*/
#include <stdio.h>
#include <memory.h>
#include "cuda_vector.h" 
 
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

 __device__  uint4 *  W;
uint32_t *d_NNonce[MAX_GPUS];
uint32_t *d_nnounce[MAX_GPUS];
__constant__  uint32_t pTarget[8];
__constant__  uint32_t key_init[16]; 
__constant__  uint32_t input_init[16];
__constant__  uint32_t  c_data[80];


#define SALSA_SMALL_UNROLL 1
#define CHACHA_SMALL_UNROLL 1
#define BLAKE2S_BLOCK_SIZE    64U 
#define BLAKE2S_OUT_SIZE      32U
#define BLAKE2S_KEY_SIZE      32U
#define BLOCK_SIZE            64U
#define FASTKDF_BUFFER_SIZE  256U
#define PASSWORD_LEN          80U
/// constants ///

static const __constant__  uint8 BLAKE2S_IV_Vec =
	{
		0x6A09E667, 0xBB67AE85, 0x3C6EF372, 0xA54FF53A,
		0x510E527F, 0x9B05688C, 0x1F83D9AB, 0x5BE0CD19
	};


static const  uint8 BLAKE2S_IV_Vechost =
{
	0x6A09E667, 0xBB67AE85, 0x3C6EF372, 0xA54FF53A,
	0x510E527F, 0x9B05688C, 0x1F83D9AB, 0x5BE0CD19
};

static const uint32_t BLAKE2S_SIGMA_host[10][16] =
{
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
};
__constant__ uint32_t BLAKE2S_SIGMA[10][16];

// Blake2S

#define BLAKE2S_BLOCK_SIZE    64U
#define BLAKE2S_OUT_SIZE      32U
#define BLAKE2S_KEY_SIZE      32U


#if __CUDA_ARCH__ >= 500
#define BLAKE_G(idx0, idx1, a, b, c, d, key) { \
idx = BLAKE2S_SIGMA[idx0][idx1]; a += key[idx]; \
    a += b; d = __byte_perm(d^a,0, 0x1032); \
	c += d; b = rotateR(b^c, 12); \
idx = BLAKE2S_SIGMA[idx0][idx1+1]; a += key[idx]; \
    a += b; d = __byte_perm(d^a,0, 0x0321); \
	c += d; b = rotateR(b^c, 7); \
} 
#else 
#define BLAKE_G(idx0, idx1, a, b, c, d, key) { \
idx = BLAKE2S_SIGMA[idx0][idx1]; a += key[idx]; \
    a += b; d = rotate(d^a,16); \
	c += d; b = rotateR(b^c, 12); \
idx = BLAKE2S_SIGMA[idx0][idx1+1]; a += key[idx]; \
    a += b; d = rotateR(d^a,8); \
	c += d; b = rotateR(b^c, 7); \
	} 
#endif

#if __CUDA_ARCH__ >= 500
#define BLAKE_G_PRE(idx0,idx1, a, b, c, d, key) { \
a += key[idx0]; \
a += b; d = __byte_perm(d^a,0, 0x1032); \
c += d; b = rotateR(b^c, 12); \
a += key[idx1]; \
a += b; d = __byte_perm(d^a,0, 0x0321); \
c += d; b = rotateR(b^c, 7); \
}
#else
#define BLAKE_G_PRE(idx0, idx1, a, b, c, d, key) { \
a += key[idx0]; \
a += b; d = rotate(d^a,16); \
c += d; b = rotateR(b^c, 12); \
a += key[idx1]; \
a += b; d = rotateR(d^a,8); \
c += d; b = rotateR(b^c, 7); \
}
#endif

#define ROTL32(x, n) ((x) << (n)) | ((x) >> (32 - (n)))
#define ROTR32(x, n) (((x) >> (n)) | ((x) << (32 - (n))))

#define BLAKE_Ghost(idx0, idx1, a, b, c, d, key) { \
idx = BLAKE2S_SIGMA_host[idx0][idx1]; a += key[idx]; \
    a += b; d = ROTR32(d^a,16); \
	c += d; b = ROTR32(b^c, 12); \
idx = BLAKE2S_SIGMA_host[idx0][idx1+1]; a += key[idx]; \
    a += b; d = ROTR32(d^a,8); \
	c += d; b = ROTR32(b^c, 7); \
		} 


static __forceinline__ __device__ void Blake2S(uint32_t * inout, const uint32_t * TheKey)
{
	uint16 V;
	uint32_t idx;
	uint8 tmpblock;
	V.hi = BLAKE2S_IV_Vec;
	V.lo = BLAKE2S_IV_Vec;
	V.lo.s0 ^= 0x01012020;
	// Copy input block for later
	tmpblock = V.lo;
	V.hi.s4 ^= BLAKE2S_BLOCK_SIZE;
	// { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	BLAKE_G_PRE(0, 1, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(2, 3, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(4, 5, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(6, 7, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(8, 9, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(10, 11, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(12, 13, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(14, 15, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	BLAKE_G_PRE(14, 10, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(4, 8, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(9, 15, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(13, 6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(1, 12, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(0, 2, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(11, 7, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(5, 3, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	BLAKE_G_PRE(11, 8, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(12, 0, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(5, 2, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(15, 13, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(10, 14, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(3, 6, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(7, 1, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(9, 4, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	BLAKE_G_PRE(7, 9, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(3, 1, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(13, 12, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(11, 14, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(2, 6, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(5, 10, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(4, 0, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(15, 8, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	BLAKE_G_PRE(9, 0, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(5, 7, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(2, 4, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(10, 15, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(14, 1, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(11, 12, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(6, 8, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(3, 13, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	BLAKE_G_PRE(2, 12, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(6, 10, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(0, 11, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(8, 3, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(4, 13, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(7, 5, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(15, 14, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(1, 9, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	BLAKE_G_PRE(12, 5, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(1, 15, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(14, 13, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(4, 10, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(0, 7, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(6, 3, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(9, 2, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(8, 11, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	BLAKE_G_PRE(13, 11, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(7, 14, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(12, 1, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(3, 9, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(5, 0, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(15, 4, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(8, 6, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(2, 10, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	BLAKE_G_PRE(6, 15, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(14, 9, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(11, 3, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(0, 8, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(12, 2, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(13, 7, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(1, 4, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(10, 5, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
	BLAKE_G_PRE(10, 2, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(8, 4, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(7, 6, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(1, 5, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(15, 11, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(9, 14, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(3, 12, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(13, 0, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	V.lo ^= V.hi;
	V.lo ^= tmpblock;
	V.hi = BLAKE2S_IV_Vec;
	tmpblock = V.lo;
	V.hi.s4 ^= 128;
	V.hi.s6 = ~V.hi.s6;
	// { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	BLAKE_G_PRE(0, 1, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(2, 3, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(4, 5, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(6, 7, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(8, 9, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(10, 11, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(12, 13, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(14, 15, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	// { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	BLAKE_G_PRE(14, 10, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(4, 8, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(9, 15, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(13, 6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(1, 12, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(0, 2, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(11, 7, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(5, 3, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	// { 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	BLAKE_G_PRE(11, 8, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(12, 0, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(5, 2, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(15, 13, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(10, 14, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(3, 6, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(7, 1, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(9, 4, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	// { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	BLAKE_G_PRE(7, 9, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(3, 1, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(13, 12, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(11, 14, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(2, 6, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(5, 10, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(4, 0, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(15, 8, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	for (int x = 4; x < 10; ++x)
	{
		BLAKE_G(x, 0x00, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
		BLAKE_G(x, 0x02, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
		BLAKE_G(x, 0x04, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
		BLAKE_G(x, 0x06, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
		BLAKE_G(x, 0x08, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
		BLAKE_G(x, 0x0A, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
		BLAKE_G(x, 0x0C, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
		BLAKE_G(x, 14, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	}
	V.lo ^= V.hi ^ tmpblock;
	((uint8*)inout)[0] = V.lo;
}


static __forceinline__ __host__ void Blake2Shost(uint32_t * inout, const uint32_t * inkey)
{
	uint16 V;
	uint32_t idx;
	uint8 tmpblock;



	V.hi = BLAKE2S_IV_Vechost;
	V.lo = BLAKE2S_IV_Vechost;
	V.lo.s0 ^= 0x01012020;

	// Copy input block for later
	tmpblock = V.lo;

	V.hi.s4 ^= BLAKE2S_BLOCK_SIZE;

	for (int x = 0; x < 10; ++x)
	{
		BLAKE_Ghost(x, 0x00, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inkey);
		BLAKE_Ghost(x, 0x02, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inkey);
		BLAKE_Ghost(x, 0x04, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inkey);
		BLAKE_Ghost(x, 0x06, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inkey);
		BLAKE_Ghost(x, 0x08, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inkey);
		BLAKE_Ghost(x, 0x0A, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inkey);
		BLAKE_Ghost(x, 0x0C, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inkey);
		BLAKE_Ghost(x, 14, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inkey);
	}

	V.lo ^= V.hi;
	V.lo ^= tmpblock;


	V.hi = BLAKE2S_IV_Vechost;
	tmpblock = V.lo;

	V.hi.s4 ^= 128;
	V.hi.s6 = ~V.hi.s6;

	for (int x = 0; x < 10; ++x)
	{
		BLAKE_Ghost(x, 0x00, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
		BLAKE_Ghost(x, 0x02, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
		BLAKE_Ghost(x, 0x04, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
		BLAKE_Ghost(x, 0x06, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
		BLAKE_Ghost(x, 0x08, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
		BLAKE_Ghost(x, 0x0A, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
		BLAKE_Ghost(x, 0x0C, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
		BLAKE_Ghost(x, 14, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	}

	V.lo ^= V.hi ^ tmpblock;

	((uint8*)inout)[0] = V.lo;
}

static __forceinline__ __device__ void fastkdf256(int thread, const uint32_t * password, uint8_t * output)
{ 

	uint8_t bufidx = 0;
	uchar4 bufhelper;
	uint8_t A[320],B[288]; 
	
 ((uintx64*)A)[0] = ((uintx64*)password)[0];
  ((uint816 *)A)[4] =  ((uint816 *)password)[0];

((uintx64*)B)[0] = ((uintx64*)password)[0];
  ((uint48 *)B)[8] = ((uint48 *)password)[0];

uint32_t input[BLAKE2S_BLOCK_SIZE/4]; uint32_t key[BLAKE2S_BLOCK_SIZE / 4]={0};

((uint816*)input)[0] = ((uint816*)input_init)[0];
((uint48*)key)[0] = ((uint48*)key_init)[0];


	for (int i = 0; i < 32; ++i)
	{

//		Blake2Stest(thread,input, key);

		bufhelper = ((uchar4*)input)[0];
		for (int x = 1; x < BLAKE2S_OUT_SIZE / 4; ++x) { bufhelper += ((uchar4*)input)[x]; }
		bufidx = bufhelper.x + bufhelper.y + bufhelper.z + bufhelper.w;
	   	
		int qbuf = bufidx/4; 
        int rbuf = bufidx&3;
		int bitbuf = rbuf << 3; 
		 uint32_t shifted[9];
  
		shift256R2(shifted, ((uint8*)input)[0], bitbuf);

		for (int k = 0; k < 9; ++k) {
			((uint32_t *)B)[k + qbuf] ^= ((uint32_t *)shifted)[k];
		}

		if (bufidx < BLAKE2S_KEY_SIZE)                          {((uint8*)B)[8] = ((uint8*)B)[0];}
		else if (bufidx > FASTKDF_BUFFER_SIZE-BLAKE2S_OUT_SIZE) {((uint8*)B)[0] = ((uint8*)B)[8];}

		if (i<31) {
		for (int k = 0; k <BLAKE2S_BLOCK_SIZE / 4; k++) {
			((uchar4*)(input))[k] = make_uchar4((A + bufidx)[4 * k], (A + bufidx)[4 * k + 1], 
                                                (A + bufidx)[4 * k + 2], (A + bufidx)[4 * k + 3]);
		}

		for (int k = 0; k <BLAKE2S_KEY_SIZE / 4; k++) {
			((uchar4*)(key))[k] = make_uchar4((B + bufidx)[4 * k], (B + bufidx)[4 * k + 1], 
                                              (B + bufidx)[4 * k + 2], (B + bufidx)[4 * k + 3]);
		} 
		Blake2S((uint32_t*)input, key);
        }
	} 
	int left = FASTKDF_BUFFER_SIZE - bufidx;
    int qleft =left/4;
    int rleft =left&3; 
	for (int k = 0; k < qleft; ++k) { ((uchar4*)output)[k] = 
make_uchar4((B + bufidx)[4 * k], (B + bufidx)[4 * k + 1], 
            (B + bufidx)[4 * k + 2], (B + bufidx)[4 * k + 3]) ^ ((uchar4*)A)[k]; }
	for (int i = 4*qleft; i < 4*qleft+rleft; ++i) { output[i] = (B + bufidx)[i] ^ A[i]; }
	for (int i = qleft*4+rleft; i < (qleft+1)*4; ++i)
		((uint8_t *)output)[i] = ((uint8_t *)B)[i - left] ^ ((uint8_t *)A)[i];
	for (int i = qleft+1; i < FASTKDF_BUFFER_SIZE/4; ++i)
		((uchar4 *)output)[i] = make_uchar4(B[4*i - left],B[4*i+1-left],
			                                B[4*i+2-left],B[4*i+3-left]) ^ ((uchar4 *)A)[i];

}

static __forceinline__ __device__ void fastkdf32( const uint32_t * password, const uint32_t * salt, uint32_t * output)
{



	uint8_t bufidx = 0;
    uchar4 bufhelper;

	uint8_t  A[320];
    uint8_t  B[288];
	// Initialize the password buffer
	((uintx64*)A)[0] = ((uintx64*)password)[0];
	((uint816*)A)[4] = ((uint816*)password)[0];
	((uintx64*)B)[0] = ((uintx64*)salt)[0];
    ((uintx64*)B)[1] = ((uintx64*)salt)[0];
uint32_t input[BLAKE2S_BLOCK_SIZE/4]; uint32_t key[BLAKE2S_BLOCK_SIZE/4]={0};
((uint816*)input)[0] = ((uint816*)password)[0];
((uint48*)key)[0] = ((uint48*)salt)[0];

	for (int i = 0; i < 32; ++i) 
	{ 

		Blake2S((uint32_t*)input, key);
		
		bufidx = 0;
		bufhelper = ((uchar4*)input)[0];
		for (int x = 1; x < BLAKE2S_OUT_SIZE / 4; ++x) { bufhelper += ((uchar4*)input)[x]; }
		bufidx = bufhelper.x + bufhelper.y + bufhelper.z + bufhelper.w;
		int qbuf = bufidx / 4;
		int rbuf = bufidx & 3;
        int bitbuf = rbuf << 3;
		uint32_t shifted[9];
		 
		shift256R2(shifted, ((uint8*)input)[0], bitbuf);

		for (int k = 0; k < 9; ++k) {
			((uint32_t *)B)[k + qbuf] ^= ((uint32_t *)shifted)[k];
		}

		if (i<31){
		if (bufidx < BLAKE2S_KEY_SIZE)                            {((uint8*)B)[8] = ((uint8*)B)[0];}
		else if (bufidx > FASTKDF_BUFFER_SIZE - BLAKE2S_OUT_SIZE) {((uint8*)B)[0] = ((uint8*)B)[8];}
//		MyUnion Test;
 
		for (uint8_t k = 0; k <BLAKE2S_BLOCK_SIZE/4 ; k++) {
	((uchar4*)(input))[k] =
	make_uchar4((A + bufidx)[4 * k], (A + bufidx)[4 * k + 1], (A + bufidx)[4 * k + 2], (A + bufidx)[4 * k + 3]);			
		}
		for (uint8_t k = 0; k <BLAKE2S_KEY_SIZE / 4; k++) {
	((uchar4*)(key))[k] =
	make_uchar4((B + bufidx)[4 * k], (B + bufidx)[4 * k + 1], (B + bufidx)[4 * k + 2], (B + bufidx)[4 * k + 3]);
		}
		}
	}
 

	uchar4 unfucked[1];
    unfucked[0] = make_uchar4(B[28 + bufidx], B[29 + bufidx],B[30 + bufidx], B[31 + bufidx]);
		 ((uint32_t*)output)[7] = ((uint32_t*)unfucked)[0] ^ ((uint32_t*)A)[7];
}

 
#define SALSA(a,b,c,d) { \
    t =a+d; b^=rotate(t,  7);    \
    t =b+a; c^=rotate(t,  9);    \
    t =c+b; d^=rotate(t, 13);    \
    t =d+c; a^=rotate(t, 18);     \
}

#define SALSA_CORE(state) { \
\
SALSA(state.s0,state.s4,state.s8,state.sc); \
SALSA(state.s5,state.s9,state.sd,state.s1); \
SALSA(state.sa,state.se,state.s2,state.s6); \
SALSA(state.sf,state.s3,state.s7,state.sb); \
SALSA(state.s0,state.s1,state.s2,state.s3); \
SALSA(state.s5,state.s6,state.s7,state.s4); \
SALSA(state.sa,state.sb,state.s8,state.s9); \
SALSA(state.sf,state.sc,state.sd,state.se); \
} 



#if __CUDA_ARCH__ >=500  
#define CHACHA_STEP(a,b,c,d) { \
a += b; d = __byte_perm(d^a,0,0x1032); \
c += d; b = rotate(b^c, 12); \
a += b; d = __byte_perm(d^a,0,0x2103); \
c += d; b = rotate(b^c, 7); \
}
#else 
#define CHACHA_STEP(a,b,c,d) { \
a += b; d = rotate(d^a,16); \
c += d; b = rotate(b^c, 12); \
a += b; d = rotate(d^a,8); \
c += d; b = rotate(b^c, 7); \
}
#endif
#define CHACHA_CORE_PARALLEL(state)	 { \
 \
    CHACHA_STEP(state.lo.s0, state.lo.s4, state.hi.s0, state.hi.s4); \
    CHACHA_STEP(state.lo.s1, state.lo.s5, state.hi.s1, state.hi.s5); \
    CHACHA_STEP(state.lo.s2, state.lo.s6, state.hi.s2, state.hi.s6); \
	CHACHA_STEP(state.lo.s3, state.lo.s7, state.hi.s3, state.hi.s7); \
	CHACHA_STEP(state.lo.s0, state.lo.s5, state.hi.s2, state.hi.s7); \
    CHACHA_STEP(state.lo.s1, state.lo.s6, state.hi.s3, state.hi.s4); \
    CHACHA_STEP(state.lo.s2, state.lo.s7, state.hi.s0, state.hi.s5); \
	CHACHA_STEP(state.lo.s3, state.lo.s4, state.hi.s1, state.hi.s6); \
\
}





static __forceinline__ __device__ uint16 salsa_small_scalar_rnd(const uint16 &X)
{
	uint16 state = X;
	uint32_t t;
	
	for (int i = 0; i < 10; ++i) { SALSA_CORE(state);}

	return(X + state);
}

static __device__ __forceinline__ uint16 chacha_small_parallel_rnd(const uint16 &X)
{ 
 
	uint16 st = X; 

	for (int i = 0; i < 10; ++i) {CHACHA_CORE_PARALLEL(st);}
	return(X + st);
}

static __device__ __forceinline__ void neoscrypt_chacha(uint16 *XV)
{

	XV[0] ^= XV[3];
	uint16 temp;
	
	XV[0] = chacha_small_parallel_rnd(XV[0]); XV[1] ^= XV[0];
  	 temp = chacha_small_parallel_rnd(XV[1]); XV[2] ^= temp;
	XV[1] = chacha_small_parallel_rnd(XV[2]); XV[3] ^= XV[1];
	XV[3] = chacha_small_parallel_rnd(XV[3]);
    XV[2] = temp;
	

}
static __device__ __forceinline__ void neoscrypt_salsa(uint16 *XV)
{

	XV[0] ^= XV[3];
	uint16 temp;
	
		XV[0] = salsa_small_scalar_rnd(XV[0]); XV[1] ^= XV[0];
		 temp = salsa_small_scalar_rnd(XV[1]); XV[2] ^= temp;
		XV[1] = salsa_small_scalar_rnd(XV[2]); XV[3] ^= XV[1];
		XV[3] = salsa_small_scalar_rnd(XV[3]);
        XV[2] = temp;

}   

 
#define SHIFT 130


__global__ __launch_bounds__(128, 1) void neoscrypt_gpu_hash_k0(int stratum, int threads, uint32_t startNonce)
{

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	int shift = SHIFT * 16 * thread;
//	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread;
		
		uint16 X[4];
		uint32_t data[80];

		for (int i = 0; i<20; i++) { ((uint4*)data)[i] = ((uint4 *)c_data)[i]; }  //ld.local.v4
		data[19] = (stratum) ? cuda_swab32(nonce) : nonce; //freaking morons !!!
		data[39] = data[19];
		data[59] = data[19];

		fastkdf256(thread,data, (uint8_t*)X);
		
			((uintx64 *)(W + shift))[0] = ((uintx64 *)X)[0];
//		((ulonglong16 *)(W + shift))[0] = ((ulonglong16 *)X)[0];
	}
}

__global__ __launch_bounds__(128, 1) void neoscrypt_gpu_hash_k01(int threads, uint32_t startNonce)
{

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	int shift = SHIFT * 16 * thread;
//	if (thread < threads)
	{


		uint16 X[4];
		((uintx64 *)X)[0]= __ldg32(&(W + shift)[0]);
	
//#pragma unroll
		for (int i = 0; i < 128; ++i)
		{			
			neoscrypt_chacha(X);
 //           ((ulonglong16 *)(W + shift))[i+1] = ((ulonglong16 *)X)[0];

			((uintx64 *)(W + shift))[i + 1] = ((uintx64 *)X)[0];
		}


	}
}

__global__ __launch_bounds__(128, 1) void neoscrypt_gpu_hash_k2(int threads, uint32_t startNonce)
{

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	int shift = SHIFT * 16 * thread;
//	if (thread < threads)
	{
		uint16 X[4];
		((uintx64 *)X)[0] = __ldg32(&(W + shift)[2048]);
		
		for (int t = 0; t < 128; t++)
		{
			int idx = X[3].lo.s0 & 0x7F;
			((uintx64 *)X)[0] ^= __ldg32(&(W + shift)[idx << 4]);
			neoscrypt_chacha(X);

		}
		((uintx64 *)(W + shift))[129] = ((uintx64*)X)[0];  // best checked

	}
}

__global__ __launch_bounds__(128, 1) void neoscrypt_gpu_hash_k3(int threads, uint32_t startNonce)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
//	if (thread < threads)
	{

		int shift = SHIFT * 16 * thread;
		uint16 Z[4];
		

		((uintx64*)Z)[0] = __ldg32(&(W + shift)[0]);

//#pragma unroll 
		for (int i = 0; i < 128; ++i)
		{
			neoscrypt_salsa(Z);
//			((ulonglong16 *)(W + shift))[i+1] = ((ulonglong16 *)Z)[0];
			((uintx64 *)(W + shift))[i + 1] = ((uintx64 *)Z)[0];
		}


	}
}

__global__ __launch_bounds__(128, 1) void neoscrypt_gpu_hash_k4(int stratum,int threads, uint32_t startNonce, uint32_t *nonceVector)
{

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
//	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread;

		int shift = SHIFT * 16 * thread;
		uint16 Z[4]; 
		uint32_t outbuf[8];

		uint32_t data[80];

		for (int i = 0; i<20; i++) { ((uint4*)data)[i] = ((uint4 *)c_data)[i]; }
		data[19] = (stratum) ? cuda_swab32(nonce) : nonce; 
		data[39] = data[19];
		data[59] = data[19];
		((uintx64 *)Z)[0] = __ldg32(&(W + shift)[2048]);
		for (int t = 0; t < 128; t++)
		{
			int idx = Z[3].lo.s0 & 0x7F; 
			((uintx64 *)Z)[0] ^= __ldg32(&(W + shift)[idx << 4]);
			neoscrypt_salsa(Z);
		}
		((uintx64 *)Z)[0] ^= __ldg32(&(W + shift)[2064]);
		fastkdf32(data, (uint32_t*)Z, outbuf);
		if (outbuf[7] <= pTarget[7]) { 
				uint32_t tmp = atomicExch(&nonceVector[0], nonce);
			}
	}
}

void neoscrypt_cpu_init(int thr_id, int threads,uint32_t *hash)
{
    
	hipMemcpyToSymbol(HIP_SYMBOL(BLAKE2S_SIGMA), BLAKE2S_SIGMA_host, sizeof(BLAKE2S_SIGMA_host), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(W), &hash, sizeof(hash), 0, hipMemcpyHostToDevice);
	hipMalloc(&d_NNonce[thr_id], sizeof(uint32_t)); 
	
} 


__host__ uint32_t neoscrypt_cpu_hash_k4(int stratum,int thr_id, int threads, uint32_t startNounce,  int order)
{
	uint32_t result[MAX_GPUS] = {0xffffffff};
	hipMemset(d_NNonce[thr_id], 0xffffffff, sizeof(uint32_t));

 
	const int threadsperblock = 128;
	
 
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);
	
 

//	neoscrypt_gpu_hash_orig << <grid, block >> >(threads, startNounce, d_NNonce[thr_id]);
	
	neoscrypt_gpu_hash_k0  << <grid, block >> >(stratum,threads, startNounce);
	neoscrypt_gpu_hash_k01 << <grid, block >> >(threads, startNounce);
	neoscrypt_gpu_hash_k2  << <grid, block >> >(threads, startNounce);
	neoscrypt_gpu_hash_k3  << <grid, block >> >(threads, startNounce);
	neoscrypt_gpu_hash_k4  << <grid, block >> >(stratum,threads, startNounce, d_NNonce[thr_id]);
	

	MyStreamSynchronize(NULL, order, thr_id);
	hipMemcpy(&result[thr_id], d_NNonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);
	
return result[thr_id];
}

__host__ void neoscrypt_setBlockTarget(uint32_t* pdata, const void *target)
{

		unsigned char PaddedMessage[80*4]; //bring balance to the force
		uint32_t input[16], key[16] = {0};
		memcpy(PaddedMessage,     pdata, 80);
		memcpy(PaddedMessage + 80, pdata, 80);
		memcpy(PaddedMessage + 160, pdata, 80);
		memcpy(PaddedMessage + 240, pdata, 80);

		((uint16*)input)[0] = ((uint16*)pdata)[0];
		((uint8*)key)[0] = ((uint8*)pdata)[0];
//		for (int i = 0; i<10; i++) { printf(" pdata/input %d %08x %08x \n",i,pdata[2*i],pdata[2*i+1]); }
		

		Blake2Shost(input,key);
		

		hipMemcpyToSymbol(HIP_SYMBOL(pTarget), target, 8 * sizeof(uint32_t), 0, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(input_init), input, 16 * sizeof(uint32_t), 0, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(key_init), key, 16 * sizeof(uint32_t), 0, hipMemcpyHostToDevice);

		hipMemcpyToSymbol(HIP_SYMBOL(c_data), PaddedMessage, 40 * sizeof(uint64_t), 0, hipMemcpyHostToDevice);
}

